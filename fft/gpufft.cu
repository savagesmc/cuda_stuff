#include "hip/hip_runtime.h"
#include "gpufft.h"

#include <memory>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include "CudaUtils.h"
#include "TimeStat.h"

using namespace std;

namespace
{
   typedef float2 GpuComplex;
}

namespace GpuUtils
{

class FftEngine::Impl
{
   bool debug_;
   int devMemSize_;
   GpuComplex *devMem_;
   hipfftHandle plan_;
public:
   Impl(int size)
   : debug_(false)
     , devMemSize_(sizeof(GpuComplex) * size)
     , devMem_(nullptr)
   {
      checkCudaErrors(hipMalloc((void **)&devMem_, devMemSize_));
      checkCudaErrors(hipfftPlan1d(&plan_, size, HIPFFT_C2C, 1));
   }

   ~Impl()
   {
      checkCudaErrors(hipfftDestroy(plan_));
      checkCudaErrors(hipFree(devMem_));
   }

   void debug(bool enableDisable)
   {
      debug_ = enableDisable;
   }

   void operator()(std::vector<Complex>& samples)
   {
      checkCudaErrors(hipMemcpy(devMem_, &samples[0], devMemSize_, hipMemcpyHostToDevice));
      checkCudaErrors(hipfftExecC2C(plan_, (hipfftComplex *)devMem_, (hipfftComplex *)devMem_, HIPFFT_FORWARD));
      checkCudaErrors(hipMemcpy(&samples[0], devMem_, devMemSize_, hipMemcpyDeviceToHost));
   }
};

FftEngine::FftEngine(int size)
: impl_(new FftEngine::Impl(size))
{ }

FftEngine::~FftEngine()
{ }

void FftEngine::debug(bool enableDisable)
{
   impl_->debug(enableDisable);
}

void FftEngine::operator()(std::vector<Complex>& samples)
{
   impl_->operator()(samples);
}

}
