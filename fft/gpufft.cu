#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/*
* Copyright 1993-2014 NVIDIA Corporation.  All rights reserved.
*
* Please refer to the NVIDIA end user license agreement (EULA) associated
* with this source code for terms and conditions that govern your use of
* this software. Any use, reproduction, disclosure, or distribution of
* this software and related documentation outside the terms of the EULA
* is strictly prohibited.
*
*/

/* Example showing the use of CUFFT for fast 1D-convolution using FFT. */

// includes, system
#include "gpufft.h"

// includes, project
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

using namespace std;

namespace GpuUtils
{

typedef float2 Complex;

vector<MyComplex> fft(const vector<MyComplex>& in)
{
   const int numSamples = in.size();
   const int mem_size = sizeof(Complex) * numSamples;

   // Allocate device memory for signal
   Complex *h_signal = (Complex *)malloc(sizeof(Complex) * numSamples);
   Complex *h_ptr = h_signal;
   vector<MyComplex>::const_iterator it = in.begin();
   vector<MyComplex>::const_iterator end = in.end();
   for (; it != end; ++it)
   {
      h_ptr->x = it->real();
      h_ptr->y = it->imag();
      ++h_ptr;
   }

   Complex *d_signal;
   Complex *d_signal2;
   checkCudaErrors(hipMalloc((void **)&d_signal, mem_size));
   checkCudaErrors(hipMalloc((void **)&d_signal2, mem_size));
   checkCudaErrors(hipMemcpy(d_signal, h_signal, mem_size, hipMemcpyHostToDevice));

   // CUFFT plan simple API
   hipfftHandle plan;
   checkCudaErrors(hipfftPlan1d(&plan, mem_size, HIPFFT_C2C, 1));
   checkCudaErrors(hipfftExecC2C(plan, (hipfftComplex *)d_signal, (hipfftComplex *)d_signal2, HIPFFT_FORWARD));

   checkCudaErrors(hipMemcpy(h_signal, d_signal2, mem_size, hipMemcpyDeviceToHost));

   vector<MyComplex> out(numSamples);
   vector<MyComplex>::iterator oit = out.begin();
   vector<MyComplex>::iterator oend = out.end();
   h_ptr = h_signal;
   for (; oit != oend; ++oit)
   {
      *oit = MyComplex(h_ptr->x, h_ptr->y);
      ++h_ptr;
   }

   // Deallocate
   checkCudaErrors(hipfftDestroy(plan));
   free(h_signal);
   checkCudaErrors(hipFree(d_signal));
   checkCudaErrors(hipFree(d_signal2));

   return out;
}

}
