// Compile with:
// nvcc --std=c++11 fft_stream.cu -o fft_stream -lcufft

#include <iostream>

#include <hip/hip_runtime.h>


#include <hipfft/hipfft.h>

#include <chrono>
#include <iomanip>

using namespace std;
using namespace std::chrono;

// Print file name, line number, and error code when a CUDA error occurs.
#define check_cuda_errors(val)  __check_cuda_errors__ ( (val), #val, __FILE__, __LINE__ )

template <typename T>
inline void __check_cuda_errors__(T code, const char *func, const char *file, int line) {
    if (code) {
    std::cout << "CUDA error at "
          << file << ":" << line << std::endl
          << "error code: " << (unsigned int) code
          << " type: \""  << hipGetErrorString(hipGetLastError()) << "\"" << std::endl
          << "func: \"" << func << "\""
          << std::endl;
    hipDeviceReset();
    exit(EXIT_FAILURE);
    }
}

int main(int argc, char *argv[]) {

    // Number of FFTs to compute.
    const int NUM_DATAs[] = {64*1024, 64*1024, 64*1024, 64*1024, 64*1024, 64*1024, 64*1024, 64*1024, 64*1024};

    // Length of each FFT.
    const int Ns[] = {128, 256, 512, 1024, 4096, 8192, 16636, 32768, 65536};

    // Number of GPU streams across which to distribute the FFTs.
    const int NUM_STREAMSs[] = { 256, 256, 256, 256, 256, 256, 256, 256, 256 };


    for (auto nn = 0; nn < 9; ++nn)
    {
       auto dbg = 0;

       auto NUM_DATA = NUM_DATAs[nn];
       auto N = Ns[nn];
       auto NUM_STREAMS = NUM_STREAMSs[nn];

       // Allocate and initialize host input data.
       float2 **h_in = new float2* [NUM_STREAMS];
       for (int ii = 0; ii < NUM_STREAMS; ii++) {
           check_cuda_errors(hipHostAlloc((void**)&h_in[ii],N*sizeof(float2), hipHostMallocWriteCombined | hipHostMallocMapped));
           for (int jj = 0; jj < N; ++jj) {
               h_in[ii][jj].x = (float) 1.f;
               h_in[ii][jj].y = (float) 0.f;
           }
       }

       // cout << dbg++ << endl;

       // Allocate and initialize host output data.
       float2 **h_out = new float2* [NUM_STREAMS];
       for (int ii = 0; ii < NUM_STREAMS; ii++) {
           check_cuda_errors(hipHostAlloc((void**)&h_out[ii],N*sizeof(float2), hipHostMallocWriteCombined | hipHostMallocMapped));
           for (int jj = 0; jj < N; ++jj) {
               h_out[ii][jj].x = 0.f;
               h_out[ii][jj].y = 0.f;
           }
       }

       // cout << dbg++ << endl;

       // Allocate pointers to device input and output arrays.
       float2 **d_in = new float2* [NUM_STREAMS];
       float2 **d_out = new float2* [NUM_STREAMS];

       // Allocate input and output arrays on device.
       for (int ii = 0; ii < NUM_STREAMS; ii++) {
           check_cuda_errors(hipHostGetDevicePointer((void**)&d_in[ii], h_in[ii], 0));
           check_cuda_errors(hipHostGetDevicePointer((void**)&d_out[ii], h_out[ii], 0));
       }

       // cout << dbg++ << endl;

       // Create CUDA streams.
       hipStream_t streams[NUM_STREAMS];
       for (int ii = 0; ii < NUM_STREAMS; ii++) {
           check_cuda_errors(hipStreamCreate(&streams[ii]));
       }

       // cout << dbg++ << endl;

       // Creates cuFFT plans and sets them in streams
       hipfftHandle* plans = (hipfftHandle*) malloc(sizeof(hipfftHandle)*NUM_STREAMS);
       for (int ii = 0; ii < NUM_STREAMS; ii++) {
           hipfftPlan1d(&plans[ii], N, HIPFFT_C2C, 1);
           hipfftSetStream(plans[ii], streams[ii]);
       }

       // cout << dbg++ << endl;

       steady_clock::time_point before = steady_clock::now();

       // Fill streams with async memcopies and FFTs.
       for (int ii = 0; ii < NUM_DATA; ii++) {
           int jj = ii % NUM_STREAMS;
           check_cuda_errors(hipMemcpyAsync(d_in[jj], h_in[jj], N*sizeof(float2), hipMemcpyHostToDevice, streams[jj]));
           hipfftExecC2C(plans[jj], (hipfftComplex*)d_in[jj], (hipfftComplex*)d_out[jj], HIPFFT_FORWARD);
           check_cuda_errors(hipMemcpyAsync(h_out[jj], d_out[jj], N*sizeof(float2), hipMemcpyDeviceToHost, streams[jj]));
       }

       // cout << dbg++ << endl;

       // Wait for calculations to complete.
       for(int ii = 0; ii < NUM_STREAMS; ii++) {
           check_cuda_errors(hipStreamSynchronize(streams[ii]));
       }

       // cout << dbg++ << endl;

       steady_clock::time_point after = steady_clock::now();

       // Free memory and streams.
       for (int ii = 0; ii < NUM_STREAMS; ii++) {
           check_cuda_errors(hipHostFree(h_in[ii]));
           // cout << ii << " " << dbg++ << endl;
           check_cuda_errors(hipHostFree(h_out[ii]));
           // cout << ii << " " << dbg++ << endl;
           /* delete[] h_in[ii]; */
           /* cout << ii << " " << dbg++ << endl; */
           /* delete[] h_out[ii]; */
           /* cout << ii << " " << dbg++ << endl; */
           check_cuda_errors(hipStreamDestroy(streams[ii]));
           // cout << ii << " " << dbg++ << endl;
       }

       // cout << dbg++ << endl;

       delete plans;

       // cout << dbg++ << endl;

       hipDeviceReset();

       // cout << dbg++ << endl;

       auto totalTime = duration<double>(after - before).count();
       auto timePer = totalTime / NUM_DATA;
       auto sampPerSec = N / timePer;

       cout << "===================================================" << endl;
       cout << "NUM_DATA:       " << NUM_DATA << endl;
       cout << "NUM_STREAMS:    " << NUM_STREAMS << endl;
       cout << "N:              " << N << endl;
       cout << "Total Time:     " << totalTime << endl;
       cout << "Time Per FFT:   " << timePer << endl;
       cout << "Samps Per Sec:  " << sampPerSec << endl;
    }

    return 0;
}
